#include "hip/hip_runtime.h"
﻿#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
// чтобы VS не ругался на __syncthreads();
//доп. инфа здесь https://devtalk.nvidia.com/default/topic/1009723/__syncthreads-and-atomicadd-are-undefined-in-visual-studio-2015/ 
#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h> 
#include ""



#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}       

const long N = 33554432; // Points count 


__global__ void calc_PI_gpu(float *x, float *y, int *totalCount) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x; // Thread id

	int threadCount = gridDim.x * blockDim.x; // Threads count to be used as a step in loop 

	int countPointsInCircle = 0;
	for (int i = idx; i < N; i += threadCount) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPointsInCircle++;
		}
	}
	atomicAdd(totalCount, countPointsInCircle); //each thread sum amount of points in circle into variable 

}


float calc_PI_CPU(float *x, float *y) {
	int countPointsInCircle = 0;
	for (int i = 0; i < N; i++) {
		if (x[i] * x[i] + y[i] * y[i] < 1) {
			countPointsInCircle++;
		}
	}
	return float(countPointsInCircle) * 4 / N;
}



int main()
{
	float *host_X, *host_Y, *gpu_X, *gpu_Y;

	host_X = (float *)calloc(N, sizeof(float));
	host_Y = (float *)calloc(N, sizeof(float));

	CUDA_CHECK_ERROR(hipMalloc((void **)&gpu_X, N * sizeof(float)));
	CUDA_CHECK_ERROR(hipMalloc((void **)&gpu_Y, N * sizeof(float)));

	hiprandGenerator_t curandGenerator; 
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT); 
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 4321ULL);
	//generate two sequences
	hiprandGenerateUniform(curandGenerator, gpu_X, N); 
	hiprandGenerateUniform(curandGenerator, gpu_Y, N);
	hiprandDestroyGenerator(curandGenerator);

	CUDA_CHECK_ERROR(hipMemcpy(host_X, gpu_X, N * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR(hipMemcpy(host_Y, gpu_Y, N * sizeof(float), hipMemcpyDeviceToHost));
	

	clock_t  start_time = clock();
	float cpu_result = calc_PI_CPU(host_X, host_Y);
	clock_t  end_time = clock();
	std::cout.precision(15);
	std::cout << "CPU time = " << (double)((end_time - start_time) * 1000 / CLOCKS_PER_SEC) << " msec" << std::endl;
	std::cout << "result: " << cpu_result << std::endl;

	float gpuTime = 0;

	hipEvent_t start;
	hipEvent_t stop;

	int blockDim = 512;
	dim3 threads(blockDim, 1);
	dim3 grid(N / (128 * blockDim), 1);
	int *total_gpu_count;
	int *host_total_gpu_count = (int *)calloc(1, sizeof(int));

	CUDA_CHECK_ERROR(hipMalloc((void **)&total_gpu_count, sizeof(int)));

	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));

	hipEventRecord(start, 0);
	calc_PI_gpu << <grid, threads >> > (gpu_X, gpu_Y, total_gpu_count);

	CUDA_CHECK_ERROR(hipMemcpy(host_total_gpu_count, total_gpu_count, sizeof(int), hipMemcpyDeviceToHost));

	int gpu_points_count = *host_total_gpu_count;
	float gpu_result = (float)gpu_points_count * 4 / N;

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "GPU time = " << gpuTime << " мсек" << std::endl;
	std::cout << "result: " << gpu_result << std::endl;

	CUDA_CHECK_ERROR(hipEventDestroy(start));
	CUDA_CHECK_ERROR(hipEventDestroy(stop));

	CUDA_CHECK_ERROR(hipFree(gpu_X));
	CUDA_CHECK_ERROR(hipFree(gpu_Y));
	CUDA_CHECK_ERROR(hipFree(total_gpu_count));

	delete host_X;
	delete host_Y;
	delete host_total_gpu_count;

	system("pause");
	return 0;
}
